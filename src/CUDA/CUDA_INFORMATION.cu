#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

int main(){
    /// retrieve some info about the CUDA device
    int nDevices;
    hipGetDeviceCount(&nDevices);
    for (int i = 0; i < nDevices; i++) {
      hipDeviceProp_t prop;
      hipGetDeviceProperties(&prop, i);
      printf("Device Number: %d\n", i);
      printf("  Device name: %s\n", prop.name);
      printf("  max Blocks Per MultiProcessor: %d\n", prop.maxBlocksPerMultiProcessor);
      printf("  max Threads Per MultiProcessor: %d\n", prop.maxThreadsPerMultiProcessor);
      printf("  max Threads Per Block: %d\n", prop.maxThreadsPerBlock);
      printf("  num SM: %d\n", prop.multiProcessorCount);
      printf("  num bytes sharedMem Per Block: %d\n", prop.sharedMemPerBlock);
      printf("  num bytes sharedMem Per Multiprocessor: %d\n", prop.sharedMemPerMultiprocessor);
      printf("  Memory Clock Rate (KHz): %d\n",
           prop.memoryClockRate);
      printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
      printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
    }

    return 0;
}