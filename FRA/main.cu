#include "hip/hip_runtime.h"
#include "./header.hpp"

int main()
{

    srand(95);
    std::vector<std::complex<double>> input;
    for (int i = 0; i < N; i++)
    {
        // Generate random numbers between -1.0 and 1.0
        double real_part = (rand() % (RAND_MAX)) / static_cast<double>(RAND_MAX) * 2.0 - 1.0;
        double imag_part = (rand() % (RAND_MAX)) / static_cast<double>(RAND_MAX) * 2.0 - 1.0;
        input.push_back(std::complex<double>(real_part, imag_part));
    }

    int grid_size = input.size() / THREAD_PER_BLOCK;
    if (input.size() % THREAD_PER_BLOCK != 0)
        grid_size++;
    cout << "ITERATIVE\n";
    std::vector<std::complex<double>> output_iterative = iterative_FF(input);

    input = permutation(input);
    int log_n = (int)(log(input.size()) / log(2)); // This will correctly return 2

    hipDoubleComplex *a; // input for GPU
    hipDoubleComplex *y; // permutation for GPU
    hipDoubleComplex *x;
    hipDoubleComplex *t;
    int *atomic_array;

    hipMallocManaged((void **)&a, sizeof(hipDoubleComplex) * (THREAD_PER_BLOCK * grid_size));
    hipMallocManaged((void **)&y, sizeof(hipDoubleComplex) * (THREAD_PER_BLOCK * grid_size));
    hipMallocManaged((void **)&x, sizeof(hipDoubleComplex) * (THREAD_PER_BLOCK * grid_size));
    hipMallocManaged((void **)&t, sizeof(hipDoubleComplex) * (THREAD_PER_BLOCK * grid_size));
    hipMallocManaged((void **)&atomic_array, sizeof(int) * (log_n + 1)); // here I have put the +1 because I need to use the 0 position as the 1st

    // Copy data from std::complex to hipDoubleComplex
    for (size_t i = 0; i < (THREAD_PER_BLOCK * grid_size); ++i)
    {
        if (i < input.size())
            a[i] = make_hipDoubleComplex(input[i].real(), input[i].imag());
        else
            a[i] = make_hipDoubleComplex(0, 0);
    }
    cout << "\nGPU_PARALLEL\n";
    cout << "grid size : " << grid_size << endl;
    cout << "Thread x Block " << THREAD_PER_BLOCK << endl;
    dim3 dimGrid(grid_size);         // 40 columns, 2 rows dsfsdf
    dim3 dimBlock(THREAD_PER_BLOCK); // 1 column, 32 rows

    parallel_fft<<<dimGrid, dimBlock>>>(N, atomic_array, a, x, t, log_n);
    hipDeviceSynchronize();

    std::vector<std::complex<double>> cuda_output_vector = cuDoubleComplexToVector(a, input.size());
    compareComplexVectors(output_iterative, cuda_output_vector);

    return 0;
}
